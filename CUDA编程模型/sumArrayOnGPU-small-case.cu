#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)
{
  const hipError_t error = call;
  if(error != hipSuccess)
  {
    printf("Error: %s:%d, ", __FILE__, __LINE__);
    printf("code: %d, reason: %s\n", error, hipGetErrorString(error));
    exit(1);
  }
}

void initialData(float *ip, int size) {
     // generate different seed for random number
     time_t t;
     srand((unsigned)time(&t));

     for(int i = 0; i < size; i++) {
             ip[i] = (float)(rand() & 0xFF)/10.0f; // 0-25.5
     }
}

void sumArraysOnHost(float *A, float *B, float *C, const int N) {
     for(int idx = 0; idx < N; idx++) {
             C[idx] = A[idx] + B[idx];
     }
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C) {
     int i = threadIdx.x;
     C[i] = A[idx] + B[idx];
}

int main(int argc, char **argv) {
    printf("%s Starting...\n", argv[0])

    // set up device
    int dev = 0;
    hipSetDevice(dev);

    // set up data size of vectors
    int nElem = 32;
    printf("Vector size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    // initialize data at host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    hipMalloc((float **)&d_A, nBytes);
    hipMalloc((float **)&d_B, nBytes);
    hipMalloc((float **)&d_C, nBytes);

    // transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    // invoke kernel at host side
    dim3 block(nElem); // 表示每个block中有多少个线程
    dim3 grid(nElem/block.x); // 表示每个grid有多少个block
    // 向量大小为32，执行配置被放在一个块内，其中包含32个元素

    sumArraysOnGPU<<<grid,block>>>(d_A, d_B, d_C);
    printf("Execution configuration <<<%d, %d>>>\n", grid.x, block.x);

    // copy kernel result back to host side
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    // add vector at host side for result checks
    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    // free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    return 0
}